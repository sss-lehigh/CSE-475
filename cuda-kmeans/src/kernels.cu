#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cfloat>
#include "kernels.h"

//// HELPER KERNELS FOR KMEANS
//////////////////////////////
/*
 * calculates the distance between a datapoint and the provided cluster
 * IN: dataset, clusters
 * OUT: membership 
 * Note: Based on the distance function used in STAMP
 */
__forceinline__ __device__ float calc_distance(
		const float* example, int nfeatures, float* cluster) {
	float dist = 0.0;
	float corr = 0.0;
	for (int i = 0; i < nfeatures; ++i) {
		float y = __fmaf_rd(example[i] - cluster[i], example[i] - cluster[i], corr);
		float t = dist + y;
		corr = y - (t - dist);
		dist = t;
	}
	return sqrtf(dist);
}

/*
 * normalizes the clusters after all thread blocks finish updating centroids
 */
__global__ void normalize_clusters(float* clusters,
		const int* __restrict__ nmembers, const int nclusters,
		const int nfeatures) {
	// find the centroid by getting average
	int p_idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (p_idx < nclusters * nfeatures) {
		int count = nmembers[p_idx / nfeatures];
		if (count < 1) {
			count = 1;
		}
		clusters[p_idx] = clusters[p_idx] / count;
	}
}

/*
 * Reset centroids for new centroid calculation
 */
__global__ void reset_clusters(float* clusters, int* nmembers,
		const int nclusters, const int nfeatures) {
	int p_idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (p_idx < nclusters * nfeatures) {
		clusters[p_idx] = 0.0;
		nmembers[p_idx / nfeatures] = 0;
	}
}
//////////////////////
//////////////////////

//// ASSIGN MEMBERSHIP
//////////////////////
// finds memberhsip for each datapoint using shared memory by copying in clusters and writing clusters 
__global__ void find_membership(const float* data,
		const int npoints, const int nfeatures, float* const clusters,
		const int nclusters, const int cchunk, int* assignments,
		int* assignments_prev, int* update) {
	extern __shared__ float s_clusters[];
	int p_idx = blockIdx.x * blockDim.x + threadIdx.x;

	// in the following
	// cluster_offset = where to start copying clusters
	// nclusters = total number of clusters
	// cchunk = how many clusters can fit in shared memory at once
	float min_dist = FLT_MAX;
	int ass = -1;
	const float limit = 0.99999;
	for (int cluster_offset = 0; cluster_offset < nclusters; cluster_offset +=
			cchunk) {
		// copy in cluster segment 
		if (threadIdx.x < cchunk && cluster_offset + threadIdx.x < nclusters) {
			for (int i = 0; i < nfeatures; ++i) {
				s_clusters[(threadIdx.x * nfeatures) + i] = clusters[(cluster_offset
						+ threadIdx.x) * nfeatures + i];
			}
		}
		__syncthreads();

		// find if new closest cluster
		if (p_idx < npoints) {
			// find assignments for the clusters in shared memory
			for (int i = 0; i < cchunk && cluster_offset + i < nclusters; ++i) {
				//printf("target cluster: %d\n", cluster_offset + i);
				float dist = calc_distance(&data[p_idx * nfeatures], nfeatures,
						&s_clusters[i * nfeatures]);
				if ((dist / min_dist) < limit) {
					min_dist = dist;
					ass = i;
				}
			}
		}
		__syncthreads();
	}

	// if assignment changed then update
	if (p_idx < npoints) {
		if (ass != -1 && ass != assignments[p_idx]) {
			++(*update);
			assignments_prev[p_idx] = assignments[p_idx];
			assignments[p_idx] = ass;
		}
	}
}

// finds membership for each datapoint using global memory only
__global__ void find_membership_global(const float* data,
		const int npoints, const int nfeatures, float* const clusters,
		const int nclusters, int* assignments, int* assignments_prev, int* update) {
	int p_idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (p_idx >= npoints) {
		return;
	}

	float min_dist = FLT_MAX;
	int ass = -1;
	const float limit = 0.99999;
	for (int i = 0; i < nclusters; ++i) {
		float dist = calc_distance(&data[p_idx * nfeatures], nfeatures,
				&clusters[i * nfeatures]);
		if ((dist / min_dist) < limit) {
			min_dist = dist;
			ass = i;
		}
	}

	if (ass != -1 && ass != assignments[p_idx]) {
		++(*update);
		assignments[p_idx] = ass;
	}
	else {
		assignments_prev[p_idx] = assignments[p_idx];
	}
}
/////////////////////////////////
/////////////////////////////////

//// UPDATE CLUSTERS [no locking]
/////////////////////////////////
__global__ void update_clusters_gmct(const float* data,
		volatile float* clusters, volatile int* nmembers, const int nclusters,
		const int npoints, const int nfeatures,
		const int* assignments,
		const int* assignments_prev, int* locks) {
	extern __shared__ float s_mem[];
	int p_idx = blockDim.x * blockIdx.x + threadIdx.x; // global target cluster
	if (p_idx >= nclusters) {
		return;
	} // if not a real cluster, than exit

	// update centroids
	for (int i = 0; i < npoints; ++i) {
		int ass = assignments[i];
		if (ass == p_idx) {
			for (int j = 0; j < nfeatures; ++j) {
				clusters[p_idx * nfeatures + j] += data[i * nfeatures + j];
			}
			++nmembers[p_idx];
		}
	}
}

__global__ void update_clusters_gmdt(const float* data,
		volatile float* clusters, volatile int* nmembers, const int nclusters,
		const int npoints, const int nfeatures,
		const int* assignments,
		const int* assignments_prev, int* locks) {
	extern __shared__ float s_mem[];
	int p_idx = blockDim.x * blockIdx.x + threadIdx.x; // global target cluster
	if (p_idx >= nclusters * nfeatures) {
		return;
	} // if not a cluster, than exit

	// update centroids
	int dim_offset = p_idx % nfeatures;
	int t_cluster = (int)(p_idx / nfeatures);
	for (int i = 0; i < npoints; ++i) {
		int ass = assignments[i];
		if (ass == t_cluster) {
			clusters[p_idx] += data[i * nfeatures + dim_offset];
			if (dim_offset == 0) {
				++nmembers[t_cluster];
			}
		}
	}
}

/*
 * SM-CT
 */
// Simple shared memory implementation, using one thread for each cluster (mimicks Source{d} implementation, maintains STAMP update approach) [cluster-centric]
__global__ void update_clusters_smct(const float* data,
		volatile float* clusters, volatile int* nmembers, const int nclusters,
		const int cchunk, const int npoints, const int nfeatures,
		const int* assignments,
		const int* assignments_prev, int* locks) {
	extern __shared__ float s_mem[];
	int p_idx = blockDim.x * blockIdx.x + threadIdx.x; // global target cluster
	if (threadIdx.x > cchunk || p_idx >= nclusters) {
		return;
	} // if not a real cluster, than exit

	float* s_clusters = reinterpret_cast<float*>(s_mem);
	int* s_nmembers = reinterpret_cast<int*>(s_clusters + (cchunk * nfeatures));

	int cluster_offset = blockIdx.x * cchunk;
	if (threadIdx.x < cchunk) {
		for (int i = 0; i < nfeatures; ++i) {
			s_clusters[threadIdx.x * nfeatures + i] = 0.0;
		}
		s_nmembers[threadIdx.x] = 0;
	}
	__syncthreads(); // needed to ensure sharedmem init is finished...

	// update centroids
	for (int i = 0; i < npoints; ++i) {
		if (p_idx >= cluster_offset && p_idx < cluster_offset + cchunk) {
			int ass = assignments[i];
			if (ass == p_idx) {
				int s_target = ass % cchunk;
				for (int j = 0; j < nfeatures; ++j) {
					s_clusters[s_target * nfeatures + j] += data[i * nfeatures + j];
				}
				++s_nmembers[s_target];
			}
		}
	}
	__syncthreads();

	// copy out
	if (p_idx >= cluster_offset && p_idx < cluster_offset + cchunk) {
		int s_target = p_idx % cchunk;
		for (int i = 0; i < nfeatures; ++i) {
			clusters[p_idx * nfeatures + i] += s_clusters[s_target * nfeatures + i];
		}
		nmembers[p_idx] = s_nmembers[s_target];
	}
}

__global__ void update_clusters_smdt(const float* data,
		volatile float* clusters, volatile int* nmembers, const int nclusters,
		const int cchunk, const int npoints, const int nfeatures,
		const int* assignments,
		const int* assignments_prev, int* locks) {
	extern __shared__ float s_mem[];
	int p_idx = blockDim.x * blockIdx.x + threadIdx.x; // global target cluster and dimension
	if (p_idx >= nclusters * nfeatures) {
		return;
	} // if not a real cluster, than exit

	float* s_clusters = reinterpret_cast<float*>(s_mem);
	int* s_nmembers = reinterpret_cast<int*>(s_clusters + (cchunk * nfeatures));

	int cluster_offset = blockIdx.x * cchunk;
	if (threadIdx.x < cchunk) {
		for (int i = 0; i < nfeatures; ++i) {
			s_clusters[threadIdx.x * nfeatures + i] = 0.0;
		}
		s_nmembers[threadIdx.x] = 0;
	}
	__syncthreads(); // needed to ensure sharedmem init is finished...

	// update centroids
	int t_cluster = p_idx / nfeatures;
	int dim_offset = p_idx % nfeatures;
	for (int i = 0; i < npoints; ++i) {
		if (t_cluster >= cluster_offset && t_cluster < cluster_offset + cchunk) {
			int ass = assignments[i];
			if (ass == t_cluster) {
				int s_target = t_cluster % cchunk;
				s_clusters[s_target * nfeatures + dim_offset] += data[i * nfeatures + dim_offset];
				if (p_idx % nfeatures == 0) {
					++s_nmembers[s_target];
				}
			}
		}
	}
	__syncthreads();

	// copy out
	if (t_cluster >= cluster_offset && t_cluster < cluster_offset + cchunk) {
		int s_target = t_cluster % cchunk;
		clusters[p_idx] += s_clusters[s_target * nfeatures + dim_offset];
		if (p_idx % nfeatures == 0) {
			nmembers[t_cluster] = s_nmembers[s_target];
		}
	}
}


// threadblock-per-centroid implementation
// Each thread block is assigned a cluster, its threads are then assigned to a feature
// No locking is needed, because there is no overlap between threads (even from different blocks)
// One thread in each tb is responsible for incrementing number of members found for its centroid
// Data is global (can be optimized to be stored read-only)
__global__ void update_clusters_shared_tb(const float* data,
		volatile float* clusters, volatile int* nmembers, const int nclusters,
		const int npoints, const int nfeatures, const int* assignments,
		const int* assignments_prev) {
	extern __shared__ float s_mem[];
	__shared__ int s_nmembers;
	float* s_cluster = reinterpret_cast<float*>(s_mem);

	// ignore unused threads
	if (threadIdx.x >= nfeatures) {
		return;
	}

	// init cluster
	s_cluster[threadIdx.x] = 0.0;
	if (threadIdx.x == 0) {
		s_nmembers = 0;
	}

	// go through all data and update cluster accordingly
	for (int i = 0; i < npoints; ++i) {
		if (assignments[i] == blockIdx.x) {
			s_cluster[threadIdx.x] += data[i * nfeatures + threadIdx.x];
			if (threadIdx.x == 0) {
				++s_nmembers;
			}
		}
	}

	// copy cluster to global memory
	clusters[blockIdx.x * nfeatures + threadIdx.x] = s_cluster[threadIdx.x];
	if (threadIdx.x == 0) {
		nmembers[blockIdx.x] = s_nmembers;
	}
}
/////////////////////////////////
